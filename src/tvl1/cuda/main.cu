#include "hip/hip_runtime.h"
#include <string>
#include <cmath>
#include <iostream>

#include <opencv2/core.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/videoio.hpp>
#include <opencv2/imgproc.hpp>

#include "tvl1.cuh"

using namespace cv;


class App {
public:
    App(const CommandLineParser& cmd);
    void initVideoSource();
    void initCuda();
    int run();
    bool isRunning() { return m_running; }
    bool doProcess() { return m_process; }
    void setRunning(bool running)      { m_running = running; }
    void setDoProcess(bool process)    { m_process = process; }
    void flowToColor(int width, int height, const float* flowData, cv::Mat& outFrame); 
protected:
    void handleKey(char key);
private:
    bool                        m_running;
    bool                        m_process;
    bool                        m_show_ui;

    int64_t                     m_t0;
    int64_t                     m_t1;
    float                       m_time;
    float                       m_frequency;

    std::string                 m_file_name;
    int                         m_camera_id;
    cv::VideoCapture            m_cap;
    cv::Mat                     m_frame;
};


App::App(const CommandLineParser& cmd)
{
    m_camera_id  = cmd.get<int>("camera");
    m_file_name  = cmd.get<std::string>("video");

    m_running    = false;
    m_process    = false;
} // ctor


void App::initCuda() {

}


void App::initVideoSource()
{
    if (!m_file_name.empty() && m_camera_id == -1)
    {
        m_cap.open(samples::findFileOrKeep(m_file_name));
        if (!m_cap.isOpened())
            throw std::runtime_error(std::string("can't open video stream: ") + m_file_name);
    }
    else if (m_camera_id != -1)
    {
        m_cap.open(m_camera_id);
        if (!m_cap.isOpened())
            throw std::runtime_error(std::string("can't open camera: ") + std::to_string(m_camera_id));
        // m_cap.set(cv::CAP_PROP_FRAME_WIDTH, 1280);
        // m_cap.set(cv::CAP_PROP_FRAME_HEIGHT, 720);
    }
    else
        throw std::runtime_error(std::string("specify video source"));
} // initVideoSource()


void App::flowToColor(int width, int height, const float* flowData, cv::Mat& outFrame) {
    cv::Mat flow_magnitude(height, width, CV_32F);
    cv::Mat flow_angle(height, width, CV_32F);

    // Populate the flow_magnitude and flow_angle matrices from the flowData array
    #pragma omp parallel for
    for (int y = 0; y < height; ++y) {
        #pragma omp simd
        for (int x = 0; x < width; ++x) {
            int index = y * width + x;
            float u = flowData[index];
            float v = flowData[index + width * height];
            flow_magnitude.at<float>(y, x) = std::sqrt(u * u + v * v);
            flow_angle.at<float>(y, x) = std::atan2(v, u);
        }
    }

    flow_magnitude = cv::min(flow_magnitude * 10.0f, 255.0f);
    flow_magnitude.convertTo(flow_magnitude, CV_8U);

    flow_angle = flow_angle * 180.0f / CV_PI / 2.0f;
    flow_angle.convertTo(flow_angle, CV_8U);

    cv::Mat hsv(height, width, CV_8UC3, cv::Scalar(0, 255, 255));

    hsv.at<cv::Vec3b>(cv::Point(0, 0))[0] = 0;
    #pragma omp parallel for
    for (int y = 0; y < height; ++y) {
        #pragma omp simd
        for (int x = 0; x < width; ++x) {
            hsv.at<cv::Vec3b>(y, x)[0] = static_cast<uint8_t>(flow_angle.at<uint8_t>(y, x));
            hsv.at<cv::Vec3b>(y, x)[1] = 255;
            hsv.at<cv::Vec3b>(y, x)[2] = flow_magnitude.at<uint8_t>(y, x);
        }
    }

    cv::cvtColor(hsv, outFrame, cv::COLOR_HSV2BGR);
}


int App::run() {
    std::cout << "Initializing..." << std::endl;

    //initCuda();
    hipDeviceProp_t devProps;
    hipGetDeviceProperties(&devProps, 0);
    std::string devName = devProps.name; 
    
    initVideoSource();

    std::cout << "Press ESC to exit" << std::endl;
    std::cout << "      'p' to toggle ON/OFF processing" << std::endl;

    m_running = true;
    m_process = true;
    m_show_ui = true;

    const int width  = static_cast<int>(m_cap.get(cv::CAP_PROP_FRAME_WIDTH));
	const int height = static_cast<int>(m_cap.get(cv::CAP_PROP_FRAME_HEIGHT));

    // buffers required for the image proccesing
    float* img = new float[width * height]{0};
    TV_L1 tvl1 = TV_L1(width, height);
    unsigned int processedFrames{0};
    double fps{0};
    cv::TickMeter timer;
    
    // Iterate over all frames
    try {
        while (isRunning()) {
            timer.reset();
            timer.start();

            m_cap.read(m_frame);

            cv::Mat m_frameGray;
            cv::cvtColor(m_frame, m_frameGray, COLOR_BGR2GRAY);

            if (m_process) {
                #pragma omp parallel for simd
                for (size_t i = 0; i < width*height; i++) {
                    img[i] = static_cast<float>(m_frameGray.data[i]);
                }
                try {
                    tvl1.runDualTVL1Multiscale(img);
                    flowToColor(width, height, tvl1.getU(), m_frameGray);
                }
                catch(const std::exception& e) {
                    std::cerr << e.what() << '\n';
                }
            }
            timer.stop();

            cv::Mat imgToShow = m_frameGray;

            std::ostringstream msg, msg2;
            fps += 1000 / timer.getTimeMilli();
            int currentFPS = 1000 / timer.getTimeMilli();
            msg << devName;
            msg2 << "FPS " << currentFPS << " (" << imgToShow.size
                << ") Time: " << cv::format("%.2f", timer.getTimeMilli()) << " msec"
                << " (process: " << (m_process ? "True" : "False") << ")";

            cv::putText(imgToShow, msg.str(), Point(10, 20), FONT_HERSHEY_SIMPLEX, 0.6, Scalar(255, 100, 0), 2);
            cv::putText(imgToShow, msg2.str(), Point(10, 50), FONT_HERSHEY_SIMPLEX, 0.6, Scalar(255, 100, 0), 2);

            if (m_show_ui) {
                try {
                    cv::imshow("Optic Flow", imgToShow);
                    int key = waitKey(1);
                    switch (key) {
                    case 27:  // ESC
                        m_running = false;
                        break;

                    case 'p':  // fallthru
                    case 'P':
                        m_process = !m_process;
                        break;

                    default:
                        break;
                    }
                }
                catch (const std::exception& e) {
                    std::cerr << "ERROR(OpenCV UI): " << e.what() << std::endl;
                    if (processedFrames > 0) {
                        delete[] img;
                        throw;
                    }
                    m_show_ui = false;  // UI is not available
                }
            }

            processedFrames++;

            if (!m_show_ui && (processedFrames > 100)) 
                m_running = false;
        }
        std::cout << std::endl;
        std::cout << "Number of frames = " << processedFrames << std::endl;
        std::cout << "Avg of FPS = " << cv::format("%.2f", fps / processedFrames) << std::endl;
    }
    catch (const std::exception& e) {
        std::cout << std::endl;
        std::cout << "Number of frames = " << processedFrames << std::endl;
        std::cout << "Avg of FPS = " << cv::format("%.2f", fps / processedFrames) << std::endl;
        delete[] img;
        return 0;
    }

    delete[] img;
    return 0;
}


int main(int argc, char** argv)
{
    const char* keys =
        "{ help h ?    |          | print help message }"
        "{ camera c    | -1       | use camera as input }"
        "{ video  v    |          | use video as input }";

    CommandLineParser cmd(argc, argv, keys);
    if (cmd.has("help"))
    {
        cmd.printMessage();
        return EXIT_SUCCESS;
    }

    try
    {
        App app(cmd);
        if (!cmd.check())
        {
            cmd.printErrors();
            return EXIT_FAILURE;
        }
        app.run();
    }
    catch (const cv::Exception& e)
    {
        std::cout << "FATAL: OpenCV error: " << e.what() << std::endl;
        return EXIT_SUCCESS;
    }
    catch (const std::exception& e)
    {
        std::cout << "FATAL: C++ error: " << e.what() << std::endl;
        return EXIT_SUCCESS;
    }

    catch (...)
    {
        std::cout << "FATAL: unknown C++ exception" << std::endl;
        return EXIT_SUCCESS;
    }

    return EXIT_SUCCESS;
} // main()
