#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <algorithm>
#include <limits>

#include "tvl1.cuh"
#include "kernels/kernels.cuh"

/**
 * Implementation of the Zach, Pock and Bischof dual TV-L1 optic flow method
 *
 * see reference:
 *  [1] C. Zach, T. Pock and H. Bischof, "A Duality Based Approach for Realtime
 *      TV-L1 Optical Flow", In Proceedings of Pattern Recognition (DAGM),
 *      Heidelberg, Germany, pp. 214-223, 2007
 *
 *
 * Details on the total variation minimization scheme can be found in:
 *  [2] A. Chambolle, "An Algorithm for Total Variation Minimization and
 *      Applications", Journal of Mathematical Imaging and Vision, 20: 89-97, 2004
 **/

TV_L1::TV_L1(int width, int height, float tau, float lambda, float theta, int nscales,
	float zfactor, int warps, float epsilon) 
{
	_width = width;
	_height = height;
	_tau = tau;
	_lambda = lambda;
	_theta = theta;
	_warps = warps;
	_epsilon = epsilon;
	_zfactor = zfactor;

    //Set the number of scales according to the size of the
    //images.  The value N is computed to assure that the smaller
    //images of the pyramid don't have a size smaller than 16x16
	const float N = 1 + std::log(std::hypot(width, height)/16.0) / std::log(1 / zfactor);
	_nscales = (N < nscales) ? N : nscales;

	_hostU = new float[2 * _width*_height];
	_hNx   = new int[_nscales];
	_hNy   = new int[_nscales];

	hipblasCreate(&_handle);

	// allocate memory for the pyramid structure
	hipMalloc(&_I0s, _nscales * _width * _height * sizeof(float));
	hipMalloc(&_I1s, _nscales * _width * _height * sizeof(float));
	hipMalloc(&_u1s, _nscales * _width * _height * sizeof(float));
	hipMalloc(&_u2s, _nscales * _width * _height * sizeof(float));
	hipMalloc(&_nx, _nscales * sizeof(int));
	hipMalloc(&_ny, _nscales * sizeof(int));
	hipMalloc(&_nxy, 2 * sizeof(int));

	hipMalloc(&_I1x, _width*_height * sizeof(float));
	hipMalloc(&_I1y, _width*_height * sizeof(float));
	hipMalloc(&_I1w, _width*_height * sizeof(float));
	hipMalloc(&_I1wx, _width*_height * sizeof(float));
	hipMalloc(&_I1wy, _width*_height * sizeof(float));
	hipMalloc(&_rho_c, _width*_height * sizeof(float));
	hipMalloc(&_v1, _width*_height * sizeof(float));
	hipMalloc(&_v2, _width*_height * sizeof(float));
	hipMalloc(&_p11, _width*_height * sizeof(float));
	hipMalloc(&_p12, _width*_height * sizeof(float));
	hipMalloc(&_p21, _width*_height * sizeof(float));
	hipMalloc(&_p22, _width*_height * sizeof(float));
	hipMalloc(&_grad, _width*_height * sizeof(float));
	hipMalloc(&_div_p1, _width*_height * sizeof(float));
	hipMalloc(&_div_p2, _width*_height * sizeof(float));
	hipMalloc(&_g1, _width*_height * sizeof(float));
	hipMalloc(&_g2, _width*_height * sizeof(float));
	hipMalloc(&_error, _width*_height * sizeof(float));

	float sigma = ZOOM_SIGMA_ZERO * std::sqrt(1.0/(_zfactor*_zfactor) - 1.0);
	sigma = std::max(sigma, PRESMOOTHING_SIGMA);
	const int bSize = (int) DEFAULT_GAUSSIAN_WINDOW_SIZE * sigma + 1;
	hipMalloc(&_B,  bSize * sizeof(float));
}

TV_L1::~TV_L1() {
	delete[] _hostU;
	delete[] _hNx;
	delete[] _hNy;
	hipblasDestroy(_handle);

	hipFree(_I0s);
	hipFree(_I1s);
	hipFree(_u1s);
	hipFree(_u2s);
	hipFree(_nx);
	hipFree(_ny);
	hipFree(_nxy);

	hipFree(_I1x);
	hipFree(_I1y);
	hipFree(_I1w);
	hipFree(_I1wx);
	hipFree(_I1wy);
	hipFree(_rho_c);
	hipFree(_v1);
	hipFree(_v2);
	hipFree(_p11);
	hipFree(_p12);
	hipFree(_p21);
	hipFree(_p22);
	hipFree(_grad);
	hipFree(_div_p1);
	hipFree(_div_p2);
	hipFree(_g1);
	hipFree(_g2);
	hipFree(_B);
	hipFree(_error);
}


/**
 * Function to compute the optical flow using multiple scales
 **/
void TV_L1::runDualTVL1Multiscale(const float *I0, const float *I1) {
	const int size = _width * _height;

	// send images to the device 
	hipMemcpy(_I0s, I0, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(_I1s, I1, size * sizeof(float), hipMemcpyHostToDevice);

	// setup initial values
	hipMemcpy(_nx, &_width, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(_ny, &_height, sizeof(int), hipMemcpyHostToDevice);
	hipMemset(_u1s + (_nscales-1 * size), 0.0f, size * sizeof(float));
	hipMemset(_u2s + (_nscales-1 * size), 0.0f, size * sizeof(float));

	// normalize the images between 0 and 255
	imageNormalization(_I0s, _I1s, _I0s, _I1s, size);

	// pre-smooth the original images
	try {
		gaussian(_I0s, _B, _nx, _ny, PRESMOOTHING_SIGMA, _I1w, &_handle);
		gaussian(_I1s, _B, _nx, _ny, PRESMOOTHING_SIGMA, _I1w, &_handle);
	}
	catch(const std::exception& e) { throw; }

	// float aux[10];
	// hipMemcpy(aux, _I0s, 10*sizeof(float), hipMemcpyDeviceToHost);
	// hipDeviceSynchronize();
	// for (size_t i = 0; i < 10; i++)
	// 	std::cout << aux[i] << ", ";
	// std::cout << std::endl;
	// return;

	// create the scales
	for (int s = 1; s < _nscales; s++)
	{
		zoomSize<<<1,1>>>(_nx + (s-1), _ny + (s-1), _nx + s, _ny + s, _zfactor);

		// zoom in the images to create the pyramidal structure
		try {
			zoomOut(_I0s + (s-1)*size, _I0s + (s*size), _B, _nx + (s-1), _ny + (s-1), _nxy, _nxy + 1, _zfactor, _I1w, _I1wx, &_handle);
			zoomOut(_I1s + (s-1)*size, _I1s + (s*size), _B, _nx + (s-1), _ny + (s-1), _nxy, _nxy + 1, _zfactor, _I1w, _I1wx, &_handle);
		}
		catch(const std::exception& e) { throw; }
	}
	hipMemcpy(_hNx, _nx, _nscales * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(_hNy, _ny, _nscales * sizeof(int), hipMemcpyDeviceToHost);

	const float invZfactor{1 / _zfactor};
	// pyramidal structure for computing the optical flow
	for (int s = _nscales-1; s > 0; s--) {
		// compute the optical flow at the current scale
		dualTVL1(_I0s + (s*size), _I1s + (s*size), _u1s + (s*size), _u2s + (s*size), _hNx[s], _hNy[s]);

		// zoom the optical flow for the next finer scale
		zoomIn(_u1s + (s*size), _u1s + (s-1)*size, _nx + s, _ny + s, _nx + (s-1), _ny + (s-1));
		zoomIn(_u2s + (s*size), _u2s + (s-1)*size, _nx + s, _ny + s, _nx + (s-1), _ny + (s-1));

		// scale the optical flow with the appropriate zoom factor
		hipblasSscal(_handle, _hNx[s-1] * _hNy[s-1], &invZfactor, _u1s + (s-1)*size, 1);
		hipblasSscal(_handle, _hNx[s-1] * _hNy[s-1], &invZfactor, _u2s + (s-1)*size, 1);
	}
	dualTVL1(_I0s, _I1s, _u1s, _u2s, _hNx[0], _hNy[0]);

	// write back to the host the result
	hipMemcpy(_hostU, _u1s, size * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(_hostU + size, _u2s, size * sizeof(float), hipMemcpyDeviceToHost);
}


/**
 *
 * Function to compute the optical flow in one scale
 *
 **/
void TV_L1::dualTVL1(const float* I0, const float* I1, float* u1, float* u2, int nx, int ny)
{
	const size_t size = nx * ny;
	const float lT = _lambda * _theta;

	centeredGradient(I1, _I1x, _I1y, nx, ny);

	// initialization of p
	hipMemset(_p11, 0.0f, size * sizeof(float));
	hipMemset(_p12, 0.0f, size * sizeof(float));
	hipMemset(_p21, 0.0f, size * sizeof(float));
	hipMemset(_p22, 0.0f, size * sizeof(float));

	const size_t TH2{THREADS_PER_BLOCK/4};
	dim3 blocks(nx / TH2 + (nx % TH2 == 0 ? 0:1), ny / TH2 + (ny % TH2 == 0 ? 0:1));
	dim3 threads(blocks.x == 1 ? nx:TH2, blocks.y == 1 ? ny:TH2);

	const size_t blocks1 = size / THREADS_PER_BLOCK + (size % THREADS_PER_BLOCK == 0 ? 0:1);
	const size_t threads1 = blocks1 == 1 ? size:THREADS_PER_BLOCK;

	for (int warpings = 0; warpings < _warps; warpings++) {
		// compute the warping of the target image and its derivatives
		bicubicInterpolationWarp<<<blocks, threads>>>(I1,  u1, u2, _I1w, nx, ny, true);
		bicubicInterpolationWarp<<<blocks, threads>>>(_I1x, u1, u2, _I1wx, nx, ny, true);
		bicubicInterpolationWarp<<<blocks, threads>>>(_I1y, u1, u2, _I1wy, nx, ny, true);

		calculateRhoGrad<<<blocks1,threads1>>>(_I1wx, _I1wy, _I1w, u1, u2, I0, _grad, _rho_c, size);

		int n{0};
		float error{INFINITY};
		while (error > _epsilon * _epsilon && n < MAX_ITERATIONS)
		{
			n++;
			// estimate the values of the variable (v1, v2)
			// (thresholding opterator TH)
			estimateThreshold<<<blocks1,threads1>>>(_rho_c, _I1wx, u1, _I1wy, u2, _grad, lT, size, _v1, _v2);

			// compute the divergence of the dual variable (p1, p2)
			divergence(_p11, _p12, _div_p1, nx ,ny);
			divergence(_p21, _p22, _div_p2, nx ,ny);

			// estimate the values of the optical flow (u1, u2)
			hipMemset(_error, 0.0f, size * sizeof(float));
			estimateOpticalFlow<<<blocks1,threads1>>>(u1, u2, _v1, _v2, _div_p1, _div_p2, _theta, size, _error);
			hipblasSasum(_handle, size, _error, 1, &error);
			error /= size;

			// compute the gradient of the optical flow (Du1, Du2)
			forwardGradient(u1, _div_p1, _v1, nx ,ny);
			forwardGradient(u2, _div_p2, _v2, nx ,ny);

			// estimate the values of the dual variable (p1, p2)
			const float taut = _tau / _theta;
			estimateGArgs<<<blocks1,threads1>>>(_div_p1, _div_p2, _v1, _v2, size, taut, _g1, _g2);

			hipblasSaxpy(_handle, size, &taut, _div_p1, 1, _p11, 1);
			hipblasSaxpy(_handle, size, &taut, _v1, 1, _p12, 1);
			hipblasSaxpy(_handle, size, &taut, _div_p2, 1, _p21, 1);
			hipblasSaxpy(_handle, size, &taut, _v2, 1, _p22, 1);

			divideByG<<<blocks1,threads1>>>(_g1, _g2, size, _p11, _p12, _p21, _p22);
		}
	}
}


/**
 *
 * Function to normalize the images between 0 and 255
 *
 **/
void TV_L1::imageNormalization(
		const float *I0,  // input image0
		const float *I1,  // input image1
		float *I0n,       // normalized output image0
		float *I1n,       // normalized output image1
		int size          // size of the image
)
{
	// obtain the max and min of each image
	int iMax0, iMax1, iMin0, iMin1;
	hipblasIsamax(_handle, size, I0, 1, &iMax0);
	hipblasIsamax(_handle, size, I1, 1, &iMax1);
	hipblasIsamin(_handle, size, I0, 1, &iMin0);
	hipblasIsamin(_handle, size, I1, 1, &iMin1);

	// obtain the max and min of both images
	float max0, max1, min0, min1;
	hipMemcpy(&max0, I0 + iMax0-1, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&max1, I1 + iMax1-1, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&min0, I0 + iMin0-1, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&min1, I1 + iMin1-1, sizeof(float), hipMemcpyDeviceToHost);

	const float max = std::max(max0, max1);
	const float min = std::min(min0, min1);
	const float den = max - min;

	if(den <= 0)
		return;

	// normalize both images
	const int blocks = size / THREADS_PER_BLOCK + (size % THREADS_PER_BLOCK == 0 ? 0:1);
	const int threads = blocks == 1 ? size : THREADS_PER_BLOCK;
	normKernel<<<blocks, threads>>>(I0, I1, I0n, I1n, min, den, size);
}


/**
 * Function to compute the divergence with backward differences
 **/
void TV_L1::divergence(
		const float *v1, // x component of the vector field
		const float *v2, // y component of the vector field
		float *div,      // output divergence
		const int nx,    // image width
		const int ny     // image height
)
{
	// compute the divergence on the central body of the image
	int blocks = (nx-1)*(ny-1) / THREADS_PER_BLOCK + ((nx-1)*(ny-1) % THREADS_PER_BLOCK == 0 ? 0:1);
	int threads = blocks == 1 ? (nx-1)*(ny-1) : THREADS_PER_BLOCK;
	bodyDivergence<<<blocks,threads>>>(v1, v2, div, nx, ny);

	// compute the divergence on the first and last rows
	blocks = (nx-1) / THREADS_PER_BLOCK + ((nx-1) % THREADS_PER_BLOCK == 0 ? 0:1);
	threads = blocks == 1 ? (nx-1) : THREADS_PER_BLOCK;
	edgeRowsDivergence<<<blocks,threads>>>(v1, v2, div, nx, ny);

	// compute the divergence on the first and last columns
	blocks = (ny-1) / THREADS_PER_BLOCK + ((ny-1) % THREADS_PER_BLOCK == 0 ? 0:1);
	threads = blocks == 1 ? (ny-1) : THREADS_PER_BLOCK;
	edgeColumnsDivergence<<<blocks,threads>>>(v1, v2, div, nx, ny);

	cornersDivergence<<<1,1>>>(v1, v2, div, nx, ny);
}


/**
 * Function to compute the gradient with forward differences
 **/
void TV_L1::forwardGradient(
		const float *f, //input image
		float *fx,      //computed x derivative
		float *fy,      //computed y derivative
		const int nx,   //image width
		const int ny    //image height
		)
{
	// compute the gradient on the central body of the image
	int blocks = (nx-1)*(ny-1) / THREADS_PER_BLOCK + ((nx-1)*(ny-1) % THREADS_PER_BLOCK == 0 ? 0:1);
	int threads = blocks == 1 ? (nx-1)*(ny-1) : THREADS_PER_BLOCK;
	bodyForwardGradient<<<blocks, threads>>>(f, fx, fy, nx, ny);

	// compute the gradient on the last row
	blocks = (nx-1) / THREADS_PER_BLOCK + ((nx-1) % THREADS_PER_BLOCK == 0 ? 0:1);
	threads = blocks == 1 ? (nx-1) : THREADS_PER_BLOCK;
	rowsForwardGradient<<<blocks, threads>>>(f, fx, fy, nx, ny);

	// compute the gradient on the last column
	blocks = (ny-1) / THREADS_PER_BLOCK + ((ny-1) % THREADS_PER_BLOCK == 0 ? 0:1);
	threads = blocks == 1 ? (ny-1) : THREADS_PER_BLOCK;
	columnsForwardGradient<<<blocks, threads>>>(f, fx, fy, nx, ny);

	// corners
	hipMemset(fx + (ny * nx - 1), 0.0f, sizeof(float));
	hipMemset(fy + (ny * nx - 1), 0.0f, sizeof(float));
}


/**
 * Function to compute the gradient with centered differences
 **/
void TV_L1::centeredGradient(
		const float* input,  //input image
		float *dx,           //computed x derivative
		float *dy,           //computed y derivative
		const int nx,        //image width
		const int ny         //image height
		)
{
	// compute the gradient on the center body of the image
	int blocks = (nx-1)*(ny-1) / THREADS_PER_BLOCK + ((nx-1)*(ny-1) % THREADS_PER_BLOCK == 0 ? 0:1);
	int threads = blocks == 1 ? (nx-1)*(ny-1) : THREADS_PER_BLOCK;
	bodyGradient<<<blocks,threads>>>(input, dx, dy, nx, ny);

	// compute the gradient on the first and last rows
	blocks = (nx-1) / THREADS_PER_BLOCK + ((nx-1) % THREADS_PER_BLOCK == 0 ? 0:1);
	threads = blocks == 1 ? (nx-1) : THREADS_PER_BLOCK;
	edgeRowsGradient<<<blocks,threads>>>(input, dx, dy, nx, ny);

	// compute the gradient on the first and last columns
	blocks = (ny-1) / THREADS_PER_BLOCK + ((ny-1) % THREADS_PER_BLOCK == 0 ? 0:1);
	threads = blocks == 1 ? (ny-1) : THREADS_PER_BLOCK;
	edgeColumnsGradient<<<blocks,threads>>>(input, dx, dy, nx, ny);

	// compute the gradient at the four corners
	cornersGradient<<<1,1>>>(input, dx, dy, nx, ny);
}


/**
 * In-place Gaussian smoothing of an image
 */
void TV_L1::gaussian(
	float* I,             // input/output image
	float* B,			  // coefficients of the 1D convolution
	const int* xdim,       // image width
	const int* ydim,       // image height
	float sigma,    // Gaussian sigma
	float* buffer,
	hipblasHandle_t* handle
)
{
	const float den  = 2*sigma*sigma;
	const float sPi = sigma * std::sqrt(M_PI * 2);
	const int   size = (int) DEFAULT_GAUSSIAN_WINDOW_SIZE * sigma + 1 ;
	int hXdim{0}, hYdim{0};
	hipMemcpy(&hXdim, xdim, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&hYdim, ydim, sizeof(int), hipMemcpyDeviceToHost);

	if (size > hXdim) {
		std::cerr << "Gaussian smooth: sigma too large." << std::endl;
		throw;
	}

	// compute the coefficients of the 1D convolution kernel
	int blocks = size / THREADS_PER_BLOCK + (size % THREADS_PER_BLOCK == 0 ? 0:1);
	int threads = (blocks == 1) ? size : THREADS_PER_BLOCK;
	convolution1D<<<blocks, threads>>>(B, size, sPi, den);

	// normalize the 1D convolution kernel
	float norm, hB;
	hipblasSasum(*handle, size, B, 1, &norm);
	hipMemcpy(&hB, B, sizeof(float), hipMemcpyDeviceToHost);
	norm = 1 / (norm * 2 - hB);
	hipblasSscal(*handle, size, &norm, B, 1);

	blocks = hYdim / THREADS_PER_BLOCK + (hYdim % THREADS_PER_BLOCK == 0 ? 0:1);
	threads = (blocks == 1) ? hYdim : THREADS_PER_BLOCK;
	// convolution of each line of the input image
    lineConvolution<<<blocks, threads>>>(I, B, xdim, ydim, size, buffer);

	blocks = hXdim / THREADS_PER_BLOCK + (hXdim % THREADS_PER_BLOCK == 0 ? 0:1);
	threads = (blocks == 1) ? hXdim : THREADS_PER_BLOCK;
	// convolution of each column of the input image
    columnConvolution<<<blocks, threads>>>(I, B, xdim, ydim, size, buffer);
}


/**
 * Downsample an image
**/
void TV_L1::zoomOut(
	const float *I,    // input image
	float* Iout,       // output image
	float* B,
	const int* nx,      // image width
	const int* ny,      // image height
	int* nxx,
	int* nyy,
	const float factor, // zoom factor between 0 and 1
	float* Is,           // temporary working image
	float* gaussBuffer,
	hipblasHandle_t* handle
)
{
	int sx, sy;
	hipMemcpy(&sx, nx, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&sy, ny, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(Is, I, sx*sy * sizeof(float), hipMemcpyDeviceToDevice);

	// compute the size of the zoomed image
	sx = (int)(sx * factor + 0.5);
	sy = (int)(sy * factor + 0.5);

	hipMemcpy(nxx, &sx, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(nyy, &sy, sizeof(int), hipMemcpyHostToDevice);

	// compute the Gaussian sigma for smoothing
	const float sigma = ZOOM_SIGMA_ZERO * std::sqrt(1.0/(factor*factor) - 1.0);

	// pre-smooth the image
	try { gaussian(Is, B, nx, ny, sigma, gaussBuffer, handle); }
	catch(const std::exception& e) { throw; }

	// re-sample the image using bicubic interpolation
	const size_t TH2{THREADS_PER_BLOCK/4};
	dim3 blocks(sx / TH2 + (sx % TH2 == 0 ? 0:1), sy / TH2 + (sy % TH2 == 0 ? 0:1));
	dim3 threads(blocks.x == 1 ? sx:TH2, blocks.y == 1 ? sy:TH2);
	bicubicResample<<<blocks, threads>>>(Is, Iout, nxx, nyy, nx, ny, factor);
}


/**
 * Function to upsample the image
**/
void TV_L1::zoomIn(
	const float *I, // input image
	float *Iout,    // output image
	const int* nx,         // width of the original image
	const int* ny,         // height of the original image
	const int* nxx,        // width of the zoomed image
	const int* nyy         // height of the zoomed image
)
{
	int sx, sy;
	hipMemcpy(&sx, nxx, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&sy, nyy, sizeof(int), hipMemcpyDeviceToHost);

	// re-sample the image using bicubic interpolation	
	const size_t TH2{THREADS_PER_BLOCK/4};
	dim3 blocks(sx / TH2 + (sx % TH2 == 0 ? 0:1), sy / TH2 + (sy % TH2 == 0 ? 0:1));
	dim3 threads(blocks.x == 1 ? sx:TH2, blocks.y == 1 ? sy:TH2);
	bicubicResample2<<<blocks, threads>>>(I, Iout, nxx, nyy, nx, ny);
}
