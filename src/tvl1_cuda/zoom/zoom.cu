#include "hip/hip_runtime.h"
// This program is free software: you can use, modify and/or redistribute it
// under the terms of the simplified BSD License. You should have received a
// copy of this license along this program. If not, see
// <http://www.opensource.org/licenses/bsd-license.html>.
//
// Copyright (C) 2012, Javier Sánchez Pérez <jsanchez@dis.ulpgc.es>
// All rights reserved.

#include <cmath>

#include "zoom.cuh"
#include "../mask/mask.cuh"
#include "../bicubic_interpolation/bicubic_interpolation.cuh"

/**
  *
  * Downsample an image
  *
**/
void zoom_out(
	const float *I,    // input image
	float *Iout,       // output image
	float* B,
	const int* nx,      // image width
	const int* ny,      // image height
	const float factor, // zoom factor between 0 and 1
	float* Is,           // temporary working image
	hipblasHandle_t* handle
)
{
	hipMemcpy(Is, I, nx*ny*sizeof(float), hipMemcpyDeviceToDevice);

	// compute the size of the zoomed image
	int nxx, nyy;
	zoom_size<<<1,1>>>(nx, ny, &nxx, &nyy, factor);

	// compute the Gaussian sigma for smoothing
	const float sigma = ZOOM_SIGMA_ZERO * std::sqrt(1.0/(factor*factor) - 1.0);

	// pre-smooth the image
	gaussian(Is, B, nx, ny, sigma, handle);

	// re-sample the image using bicubic interpolation
	#pragma omp parallel for
	for (int i1 = 0; i1 < nyy; i1++)
		for (int j1 = 0; j1 < nxx; j1++) {
			const float i2  = (float) i1 / factor;
			const float j2  = (float) j1 / factor;

			float g = bicubic_interpolation_at(Is, j2, i2, nx, ny, false);
			Iout[i1 * nxx + j1] = g;
		}
}


/**
  *
  * Function to upsample the image
  *
**/
void zoom_in(
	const float *I, // input image
	float *Iout,    // output image
	int nx,         // width of the original image
	int ny,         // height of the original image
	int nxx,        // width of the zoomed image
	int nyy         // height of the zoomed image
)
{
	// compute the zoom factor
	const float factorx = ((float)nxx / nx);
	const float factory = ((float)nyy / ny);

	// re-sample the image using bicubic interpolation
	#pragma omp parallel for
	for (int i1 = 0; i1 < nyy; i1++)
	for (int j1 = 0; j1 < nxx; j1++)
	{
		float i2 =  (float) i1 / factory;
		float j2 =  (float) j1 / factorx;

		float g = bicubic_interpolation_at(I, j2, i2, nx, ny, false);
		Iout[i1 * nxx + j1] = g;
	}
}


/**
  *
  * Compute the size of a zoomed image from the zoom factor
  *
**/
__global__ void zoom_size(
	const int* nx,      // width of the orignal image
	const int* ny,      // height of the orignal image
	int* nxx,    // width of the zoomed image
	int* nyy,    // height of the zoomed image
	float factor // zoom factor between 0 and 1
)
{
	//compute the new size corresponding to factor
	//we add 0.5 for rounding off to the closest number
	*nxx = (int)((float) *nx * factor + 0.5);
	*nyy = (int)((float) *ny * factor + 0.5);
}